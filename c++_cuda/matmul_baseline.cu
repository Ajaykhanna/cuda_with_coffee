#include <hip/hip_runtime.h>

#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <assert.h>

// Vector addition with CUDA Kernel
__global__ void matMul(int* a, int* b, int* c, int n) {
	// Compute each thread's row
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	// Compute each thread's cols
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	int temp_sum = 0;
	// Sanity Check: Guarding Vector Bounday

	if ((row < n ) && (col < n)){
		// Iterate over rows (l --> r) and cols (u --> d) 
		for (int k = 0; k < n; k++){
			// Store results for a singe element
			temp_sum += a[row * n + k] * b[k * n + col];
		}
		// Assign result
		c[row * n + col] = temp_sum;
	}
}

// Using Random Number generator to generate matrix elements of size n
// and integers between 0 to 99
void rand_matrix(int* a, int n){
	for (int i = 0; i < n; i++){
		for (int j = 0; j < n; j++) {
		a[i * n + j] = rand() % 100;
		}
	}
}

// Verify vector addition results
void verify_results(int* a, int* b, int* c, int n){
	int *verify_c;
	verify_c = (int*)malloc(n * n * sizeof(int));
	int temp_sum;
	for (int i = 0; i < n; i++){
		for (int j = 0; j < n; j++){
			temp_sum = 0; 
			for (int k = 0; k < n; k++){
				temp_sum += a[i * n + k] * b[k * n + j];
			}
			verify_c[i * n + j] = temp_sum;
		}
	}
	for (int i = 0; i < n; i++){
		for (int j = 0; j < n; j++){
			assert(c[i * n + j] == verify_c[i * n + j]);
		}
	}

}

int main(){
	// Vector size of 2^10 (1024 X 1024 elements)
	int n = 1 << 10;
	size_t bytes = n * n * sizeof(int);
	
	// Host Vector Pointers
	int *h_a, *h_b, *h_c;
	
	// Device vector pointers
	int *d_a, *d_b, *d_c;

	//Allocate host memory
	h_a = (int*)malloc(bytes);
	h_b = (int*)malloc(bytes);
	h_c = (int*)malloc(bytes);

	// Allocate device memory
	hipMalloc(&d_a, bytes);
	hipMalloc(&d_b, bytes);
	hipMalloc(&d_c, bytes);
	
	// Generate random matrix elements for a and b matrices
	rand_matrix(h_a, n);
	rand_matrix(h_b, n);

	// Copy data from Host to Device
	hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice);
	hipMemcpy(d_b, h_b, bytes, hipMemcpyHostToDevice);

	// Threads per block
	int BLOCK_SIZE = 16;
	
	// Blocks in each dimension (No padding)
	int GRID_SIZE = (int) ceil(n / BLOCK_SIZE);

	// Use dim3 objects
	dim3 grid(GRID_SIZE, GRID_SIZE);
	dim3 threads(BLOCK_SIZE, BLOCK_SIZE);

	// Launch GPU Kernel/Function on default stream w/o sharedMem
	matMul<<<grid, threads>>>(d_a, d_b, d_c, n);

	// Copy sum array c from device to host
	hipMemcpy(h_c, d_c, bytes, hipMemcpyDeviceToHost);
	
	// Check Results for errors
	verify_results(h_a, h_b, h_c, n);

//	for (int i = 0; i < int(10); i++){
//		for (int j = 0; j < int(10); j++){
//			printf(h_a[i][j]);
//		}
//		printf("The sum of A vector element %d + and B vector element %d is = %d \n", h_a[i], h_b[i], h_c[i]);
//	}
	printf("TASK DONE SUCCESSFULLY\n");

	// Free host memory
	free(h_a);
	free(h_b);
	free(h_c);
	
	// Free memory on device
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
	
	return 0;


}
